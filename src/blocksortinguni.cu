#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << host_data[i * n + j] << "\t";
		}
		std::cout << "\n";
	}

}

//__global__ void bitonic_sort_step(uint *dev_values, int k, int p, int n) {
__global__ void block_sorting(uint *d_vec, int n) {

	uint i = blockDim.x * blockIdx.x + threadIdx.x;

	for (int k = 2; k <= BLOCK_SIZE; k <<= 1) { // sorting only block size row

		for (int p = k >> 1; p > 0; p = p >> 1) {

			uint ixp = i ^ p;

			/* The threads with the lowest ids sort the array. */
			if (i < ixp) {

				bool up = ((threadIdx.x & k) == 0); // sorting only block size matrix row

				// Sort ascending or descending according to up value
				if ((d_vec[i] > d_vec[ixp]) == up) {
					// exchange(i,ixj);
					uint temp = d_vec[i];
					d_vec[i] = d_vec[ixp];
					d_vec[ixp] = temp;
				}

			}

			__syncthreads();
		}
	}
}

int main(int argc, char** argv) {
	uint num_of_elements;
	scanf("%d", &num_of_elements);
	int n = num_of_elements;
	int m = num_of_elements;
	uint mem_size = sizeof(int) * (n * m);
	uint *h_vec = (uint *) malloc(mem_size);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < m; j++) {
			scanf("%d", &h_vec[i * n + j]);
		}
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec;

	cudaTest(hipMalloc((void **) &d_vec, mem_size));

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size, hipMemcpyHostToDevice));

		dim3 dimBlock(BLOCK_SIZE, 1);
		dim3 dimGrid((n*m - 1) / dimBlock.x + 1, 1);

		hipEventRecord(start);
		block_sorting<<<dimGrid, dimBlock>>>(d_vec, n);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec, mem_size, hipMemcpyDeviceToHost);

	hipFree(d_vec);

	if (ELAPSED_TIME != 1) {
		print(h_vec, n, m);
	}

	free(h_vec);

	return 0;
}

/*
 * for (int p = 0; p < logn; p++) {
 for (int q = 0; q <= p; q++) {

 int d = 1 << (p-q);
 //for(int i = 0; i < n; i++) {
 bool up = ((col >> p) & 2) == 0;

 if ((col & d) == 0 && (As[row][col] > As[row][col | d]) == up) {
 int t = As[row][col];
 As[row][col] = As[row][col | d];
 As[row][col | d] = t;
 }
 //			}
 }
 }
 */
