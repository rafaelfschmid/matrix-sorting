#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif


void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

// Matrix multiplication kernel called by MatMul()
__global__ void block_sorting(uint* A, int n, int m) {
// Each thread computes one element of C
// by accumulating results into Cvalue
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];

	C.elements[row * C.width + col] = Cvalue;

}


int main(int argc, char** argv) {

	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_elements);
	uint mem_size = sizeof(int) * (num_of_elements * num_of_elements);
	uint *h_vec = (uint *) malloc(mem_size);
	for (i = 0; i < num_of_elements; i++) {
		for (i = 0; i < num_of_elements + 1; i++) {
			scanf("%d", &h_vec[i]);
		}
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));

		hipEventRecord(start);
		block_sorting(d_vec, num_of_elements, num_of_elements);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	hipFree(d_vec);

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_vec);

	return 0;
}

