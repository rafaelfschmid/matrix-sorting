void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {

	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(int) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(int) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	uint *h_value = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
		h_value[i] = i;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_value, *d_value_out, *d_vec, *d_vec_out;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value_out, mem_size_vec));

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
		cudaTest(hipMemcpy(d_value, h_value, mem_size_vec, hipMemcpyHostToDevice));

		hipEventRecord(start);
		uint threadCount = block_sorting(d_vec_out, d_value_out, d_vec, d_value, 1, num_of_elements, 1);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	for (i = 0; i < num_of_segments; i++) {
		for (uint j = h_seg[i]; j < h_seg[i + 1]; j++) {
			uint segIndex = i << mostSignificantBit;
			h_vec[j] -= segIndex;
		}
	}

	hipFree(d_vec);
	hipFree(d_vec_out);
	hipFree(d_value);
	hipFree(d_value_out);

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_value);

	return 0;
}

